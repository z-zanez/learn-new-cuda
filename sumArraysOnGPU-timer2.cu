#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if(error != hipSuccess) \
    { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

double cpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;
    for(int i = 0; i < N; i++)
    {
        if(abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if(match)
    {
        printf("Arrays match.\n\n");
    }
}

void initialData(float *ip, int size)
{
    time_t t;
    srand((unsigned int) time(&t));
    for(int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for(int idx = 0; idx < N; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void sumArraysOnGPU2(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = 2 * i;
    if (idx < n)
    {
        C[idx] = A[idx] + B[idx];
        if (idx + 1 < n)
        {
            C[idx + 1] = A[idx + 1] + B[idx + 1];
        }
    }
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 1 << 24;
    printf("Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    double iStart, iElaps;

    // initialize data at host side
    iStart = cpuSecond();
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    iElaps = cpuSecond() - iStart;

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add vector at host side for result checks
    iStart = cpuSecond();
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    iElaps = cpuSecond() - iStart;

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float **)&d_A, nBytes));
    CHECK(hipMalloc((float **)&d_B, nBytes));
    CHECK(hipMalloc((float **)&d_C, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int block_size = 256;
    dim3 block(block_size);
    dim3 grid((nElem + block.x * 2 - 1) / (block.x * 2));

    iStart = cpuSecond();
    sumArraysOnGPU2<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipDeviceSynchronize());
    iElaps = cpuSecond() - iStart;
    printf("sumArraysOnGPU2 <<<  %d, %d  >>> elapsed %f sec\n", grid.x, block.x, iElaps);

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}
