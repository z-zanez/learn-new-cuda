#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if(error != hipSuccess) \
    { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

void initialInt(int *ip, int size)
{
    time_t t;
    srand((unsigned int) time(&t));
    for(int i = 0; i < size; i++)
    {
        ip[i] = (int)(rand() & 0xFF);
    }
}

void printMatrix (int *C, const int nx, const int ny)
{
    int *ic = C;
    printf("\nMatrix: (%d.%d)\n", nx, ny);
    for(int iy = 0; iy < ny; iy++)
    {
        for(int ix = 0; ix < nx; ix++)
        {
            printf("%3d ", ic[ix]);
        }
        ic += nx;
        printf("\n");
    }
    printf("\n");
}

__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;
    printf("Thread_id (%d, %d) block_id (%d, %d) coordinate (%d, %d) global index %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // get device information
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set matrix dimension
    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(int);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    int *h_A;
    h_A = (int *)malloc(nBytes);

    // initialize host matrix with integer
    initialInt(h_A, nxy);
    printMatrix(h_A, nx, ny);

    // malloc device memory
    int *d_MatA;
    CHECK(hipMalloc((int**)&d_MatA, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));

    // set up execution configuration
    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // invoke the kernel
    printThreadIndex<<<grid, block>>>(d_MatA, nx, ny);
    CHECK(hipDeviceSynchronize());

    // free host and device memory
    CHECK(hipFree(d_MatA));
    free(h_A);

    // reset device
    CHECK(hipDeviceReset());
    return(0);
}